#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define TILE_WIDTH 16 

extern "C" void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int width);
// <<< |||
extern "C" void gpu_mat_mul_new(float* h_M, float* h_N, float* h_P, int height, int dummy, int width);
// >>>

__global__
void gpu_mat_mul_kernel(float* M, float* N, float* P, int width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  // Each thread works on an element of P
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  int phase_num = ceil(width / (float)TILE_WIDTH);
  float Pvalue = 0;
  // Each thread loads 'Row'th row of M and 'Col'th column of N
  for (int ph = 0; ph < phase_num; ++ph) {
        
    Mds[ty][tx] = M[Row * width + ph * TILE_WIDTH + tx];   
    Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + Col];

    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) { 
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }

    __syncthreads();
  }

  P[Row * width + Col] = Pvalue;
}

// <<< |||
__global__
void gpu_mat_mul_kernel_new(float* M, float* N, float* P, int height, int dummy, int width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = bx * TILE_WIDTH + tx;
  int Col = by * TILE_WIDTH + ty;

  int phase_num = ceil(dummy / (float)TILE_WIDTH);
  float Pvalue = 0.0;

  for (int ph = 0; ph < phase_num; ++ph) {

    if (Row < height && ph * TILE_WIDTH + ty < dummy)
      Mds[tx][ty] = M[Row * dummy + ph * TILE_WIDTH + ty];
    if (Col < width && ph * TILE_WIDTH + tx < dummy)
      Nds[tx][ty] = N[(ph * TILE_WIDTH + tx) * width + Col];

    __syncthreads();
    
    for (int k = 0; k < TILE_WIDTH; ++k)
      if (ph * TILE_WIDTH + k < dummy)
        Pvalue += Mds[tx][k] * Nds[k][ty];

    __syncthreads();

    if (Row < height && Col < width)
      P[Row * width + Col] = Pvalue;
  }
}
// >>>

void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int width) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = width * width * size_of_float;
  size_t size_N = width * width * size_of_float;
  size_t size_P = width * width * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim(ceil(width / (float)(TILE_WIDTH)), ceil(width / (float)(TILE_WIDTH)), 1);
  dim3 block_dim(TILE_WIDTH, TILE_WIDTH,1);
  gpu_mat_mul_kernel<<<grid_dim, block_dim>>>(d_M, d_N, d_P, width);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  // Free device memory for M, N, P
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

// <<< |||
void gpu_mat_mul_new(float* h_M, float* h_N, float* h_P, int height, int dummy, int width) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = height * dummy * size_of_float;
  size_t size_N = dummy * width * size_of_float;
  size_t size_P = height * width * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim(ceil(height / (float)(TILE_WIDTH)), ceil(width / (float)(TILE_WIDTH)), 1);
  dim3 block_dim(TILE_WIDTH, TILE_WIDTH,1);
  gpu_mat_mul_kernel_new<<<grid_dim, block_dim>>>(d_M, d_N, d_P, height, dummy, width);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}
/// >>>
